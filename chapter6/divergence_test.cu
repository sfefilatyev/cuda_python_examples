#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void divergence_test_ker()
{
    if(threadIdx.x % 2 == 0)
        printf("threadIdx.x %d : This is an even thread.\n", threadIdx.x);
    else
        printf("threadIdx.x %d : This is an odd thread.\n", threadIdx.x);
}

__host__ int main()
{
    hipSetDevice(0);
    divergence_test_ker <<<1, 32>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
}
